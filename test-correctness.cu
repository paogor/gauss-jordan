#include "hip/hip_runtime.h"
#include<iostream>
#include<vector>
#include <algorithm>

#include <boost/math/special_functions/next.hpp>

#include<thrust/device_vector.h>
#include<thrust/host_vector.h>

#include"gauss-jordanCPU.hpp"
#include"gauss-jordan0.hpp"
#include"gauss-jordan1.hpp"
#include"gauss-jordan2.hpp"
#include"gauss-jordan3.hpp"
#include"gauss-jordan4.hpp"
#include"gauss-jordan5.hpp"

using std::vector;
using thrust::device_vector;
using thrust::host_vector;
using thrust::raw_pointer_cast;

using boost::math::float_distance;

template <typename T>
std::ostream& operator<<(std::ostream& os, std::vector<T> A){

  for(size_t i=0; i < A.size(); ++i)
      os<<A[i]<<" ";
  
  return os;

}

template <typename T>
std::ostream& operator<<(std::ostream& os, thrust::host_vector<T> A){

  for(size_t i=0; i < A.size(); ++i)
      os<<A[i]<<" ";
  
  return os;

}

template <typename T>
std::vector<T> gmmm(size_t n, T * A, T * B )
{

  std::vector<T> C(n*n,0);

  for(size_t i=0; i <n; ++i)
    for(size_t j=0; j <n; ++j)
      for(size_t k=0; k< n; ++k)
        C[(i*n)+j] += A[(i*n)+k]*B[(k*n)+j];

  return C;

}


int main(){

  const int n = 4;
  vector<double> A(n*n);

//  std::generate(A.begin(), A.end(), std::rand);
//  double max = *std::max_element(A.begin(), A.end());

  for(int i = 0; i<n; ++i)
   for(int j = 0; j<n; ++j)
     A[(i*n)+i] = 1 ;
 
  std::cout<<A<<std::endl;

  vector<double> ACPU = A;
  host_vector<double> h_A0, h_A1, h_A2, h_A3, h_A4, h_A5;

  device_vector<double> d_A0 = A;
  device_vector<double> d_A1 = A;
  device_vector<double> d_A2 = A;
  device_vector<double> d_A3 = A;
  device_vector<double> d_A4 = A;
  device_vector<double> d_A5 = A;

  gauss_jordanCPU(n, ACPU.data());

  gauss_jordan0<<<1,n,n*n*sizeof(double)>>>(n, raw_pointer_cast(d_A0.data()));
  h_A0 = d_A0;

  gauss_jordan1<<<1,n,n*n*sizeof(double)>>>(n, raw_pointer_cast(d_A1.data()));
  h_A1 = d_A1;

  gauss_jordan2<<<1,n,(n*n+1)*sizeof(double)>>>(n, raw_pointer_cast(d_A2.data()));
  h_A2 = d_A2;
 
  gauss_jordan3<<<dim3(1,1,1),dim3(n,n,1),(n*n)*sizeof(double)>>>(n, raw_pointer_cast(d_A3.data()));
  h_A3 = d_A3;

  gauss_jordan4<<<dim3(1,1,1),dim3(n,n,1),(n*n+1)*sizeof(double)>>>(n, raw_pointer_cast(d_A4.data()));
  h_A4 = d_A4;

  gauss_jordan5<<<1,2*n,n*n*2*sizeof(double)>>>(n, raw_pointer_cast(d_A5.data()));
  h_A5 = d_A5;

/*  double total_distance = 0.0;

  for(int i = 0; i<ACPU.size(); ++i){
    std::cout<<  float_distance(ACPU[i],h_A0[i])<<std::endl;
      std::cout<< float_distance(ACPU[i],h_A1[i])<<std::endl;
      std::cout<< float_distance(ACPU[i],h_A2[i])<<std::endl;
      std::cout<< float_distance(ACPU[i],h_A3[i])<<std::endl;
      std::cout<< float_distance(ACPU[i],h_A4[i])<<std::endl;
      std::cout<< float_distance(ACPU[i],h_A5[i])<<std::endl;
  }




  std::cout<<gmmm(n,A.data(),ACPU.data())<<std::endl
           <<gmmm(n,A.data(),h_A0.data())<<std::endl
           <<gmmm(n,A.data(),h_A1.data())<<std::endl
           <<gmmm(n,A.data(),h_A2.data())<<std::endl
           <<gmmm(n,A.data(),h_A3.data())<<std::endl
           <<gmmm(n,A.data(),h_A4.data())<<std::endl
           <<gmmm(n,A.data(),h_A5.data())<<std::endl;


 // std::cout<<total_distance<<std::endl;
*/

  std::cout<<h_A5<<std::endl;

  return 0;
}
